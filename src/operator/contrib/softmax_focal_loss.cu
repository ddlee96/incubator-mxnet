#include "hip/hip_runtime.h"
// Modified from Official Caffe2 implementation
// Author: ddlee, me@ddlee.cn

#include "./softmax_focal_loss-inl.h"
#include <mshadow/tensor.h>
#include "../mshadow_op.h"

namespace mshadow {
namespace cuda {

    template<typename DType>
    __global__ void SoftmaxKernel(const int N, const int A,
        const int num_classes, const DType *Xdata, DType *Pdata) {
        CUDA_KERNEL_LOOP(index, N * A) {
        int D = num_classes * A;
        int a = index % A;
        int i = index / A;

        // Subtract max on each cell for numerical reasons
        float max_val = -FLT_MAX;
        for(int c = a * num_classes; c < (a + 1) * num_classes; ++c) {
            int idx = i * D +  c;
            max_val = max(max_val, Xdata[idx]);
        }
        // Exponentiate
        float expsum = 0.0f;
        for(int c = a * num_classes; c < (a + 1) * num_classes; ++c) {
            int idx = i * D + c;
            float expx = exp(Xdata[idx] - max_val);
            Pdata[idx] = expx;
            expsum += expx;
        }
        // Normalize
        for(int c = a * num_classes; c < (a + 1) * num_classes; ++c) {
            int idx = i * D + c;
            Pdata[idx] /= expsum;
        }
        }
    }

    template<typename DType>
    __global__ void SoftmaxFocalLossKernel(
        const int N, const int A, const int num_classes,
        const DType *Pdata, const DType *targets, DType *losses,
        const int valid_cnt, const float gamma, const float alpha) {
        CUDA_KERNEL_LOOP(i, N * A) {
        int D = A * num_classes;
        int a = i % A;
        int n = i / A;
        const int label = static_cast<int>(targets[i]);

        float Np = max(valid_cnt, 1);
        float z = (label == 0) * (1 - alpha) / Np +
                    (label >= 1) * alpha / Np;

        losses[i] = 0.0;
        if (label >= 0) {
            int offset = a * num_classes;
            int idx = n * D + offset + label;
            losses[i] =
                -(pow(1.0 - Pdata[idx], gamma) *
                log(max(Pdata[idx], FLT_MIN))) * z;
        }
        }
    }

    template<typename DType>
    __global__ void SoftmaxFocalLossGradientWeightKernel(
        const int N, const int A, const int num_classes,
        const DType *Pdata, const DType *targets, DType *buff,
        const int valid_cnt, const float gamma, const float alpha) {
        CUDA_KERNEL_LOOP(i, N * A) {
        int D = A * num_classes;
        int a = i % A;
        int n = i / A;
        const int label = static_cast<int>(targets[i]);
        float Np = max(valid_cnt, 1);
        float z =  (label == 0) * (1 - alpha) / Np +
                    (label >= 1) * alpha / Np;

        buff[i] = 0.0;
        if (label >= 0) {
            int offset = a * num_classes;
            int idx = n * D + offset + label;
            float onemp = 1. - Pdata[idx];
            float p = Pdata[idx];
            buff[i] =
                (-pow(onemp, gamma) +
                gamma * pow(onemp, gamma - 1) * p * log(max(p, FLT_MIN))) * z;
        }
        }
    }

    template<typename DType>
    __global__ void SoftmaxFocalLossGradientKernel(
        const int N, const int D, const int num_classes,
        const DType *Pdata, const DType *targets, const DType *buff,
        const DType *d_loss_data, DType *dX) {
        CUDA_KERNEL_LOOP(i, N * D) {
        int A = D / num_classes;
        int d = i % D;
        int a = d / num_classes;
        int c = d % num_classes;
        int n = i / D;
        float d_loss = *d_loss_data;

        int ind = n * A + a;
        const int label = static_cast<int>(targets[ind]);

        float c1 = (label >= 0) * 1.0;
        float c2 = (label == c) * 1.0;
        dX[i] = 0.0;
        dX[i] = c1 * d_loss * buff[ind] * (c2 - Pdata[i]);
        }
    }

    template<typename DType>
    inline void SoftmaxFocalLossForward(const Tensor<gpu, 3, DType> &X, // Logits; data
                                        const Tensor<gpu, 2, DType> &T, // Labels; labels
                                        const Tensor<gpu, 2, DType> &loss,
                                        const Tensor<gpu, 3, DType> &P, //softmax probability, going to be re-used in gradient; prob
                                        const int valid_cnt_,
                                        const float gamma_,
                                        const float alpha_) {
        int N = X.size(0); // batch
        int num_classes = X.size(1); // num of class
        int A = X.size(2); // third dimension, num of anchor in detection case

        const DType *Xdata = X.dptr_;

        DType *Pdata = P.dptr_;

        // Softmax Kernel
        dim3 dimGrid(N * A);
        dim3 dimBlock(kMaxThreadsPerBlock);
        CheckLaunchParam(dimGrid, dimBlock, "SoftmaxKernel");
        // calculate softmax probabilities: Pdata
        hipStream_t stream = Stream<gpu>::GetStream(loss.stream_);
        SoftmaxKernel<DType><<<dimGrid, dimBlock, 0, stream>>>(
            N, A, num_classes, Xdata, Pdata);

        // Compute loss for each x,y location
        const DType *Tdata = T.dptr_;
        DType *Ldata = loss.dptr_;


        CheckLaunchParam(dimGrid, dimBlock, "SoftmaxFocalLossKernel");
        // hipStream_t stream = Stream<gpu>::GetStream(losses_.stream_);
        SoftmaxFocalLossKernel<DType><<<dimGrid, dimBlock, 0, stream>>>(
            N, A, num_classes, Pdata, Tdata, Ldata, valid_cnt_, gamma_, alpha_);

    }


    template<typename DType>
    inline void SoftmaxFocalLossBackwardAcc(const Tensor<gpu, 3, DType> &X, // Logits; data
                                            const Tensor<gpu, 2, DType> &T, // Labels; labels
                                            const Tensor<gpu, 3, DType> &P, //softmax probability; prob
                                            const Tensor<gpu, 3, DType> &dX, // gradient in
                                            const Tensor<gpu, 2, DType> &dloss, // gradient out
                                            const Tensor<gpu, 2, DType> &buff_, // aux buff_ Tensor
                                            const int valid_cnt_,
                                            const float gamma_,
                                            const float alpha_) {
        int N = X.size(0);
        int num_classes = X.size(1);
        int A = X.size(2);

        const DType *Tdata = T.dptr_;
        const DType *Pdata = P.dptr_;

        DType *Bdata = buff_.dptr_;

        // Compute the weight for gradients
        dim3 dimGrid(N * A);
        dim3 dimBlock(kMaxThreadsPerBlock);
        CheckLaunchParam(dimGrid, dimBlock, "SoftmaxFocalLossGradientWeightKernel");
        hipStream_t stream = Stream<gpu>::GetStream(dX.stream_);
        SoftmaxFocalLossGradientWeightKernel<DType><<<dimGrid, dimBlock, 0, stream>>>(N, A, num_classes, Pdata, Tdata, Bdata,
            valid_cnt_, gamma_, alpha_);

        int D = A * num_classes;
        DType *dXdata = dX.dptr_;
        DType *dLdata = dloss.dptr_;
        
        // Compute the gradient with the weights
        dim3 dimGrid_(N * D);
        CheckLaunchParam(dimGrid_, dimBlock, "SoftmaxFocalLossGradientWeightKernel");
        SoftmaxFocalLossGradientKernel<DType><<<dimGrid_, dimBlock, 0, stream>>>(N, D, num_classes, Pdata, Tdata, Bdata, dLdata, dXdata);

    }
} // cuda


    template<typename DType>
    inline void SoftmaxFocalLossForward(const Tensor<gpu, 3, DType> &X, // Logits; data
                                        const Tensor<gpu, 2, DType> &T, // Labels; labels
                                        const Tensor<gpu, 2, DType> &loss, // aux losses_ Tensor
                                        const Tensor<gpu, 3, DType> &P, //softmax probability, going to be re-used in gradient; prob
                                        const int valid_cnt_,
                                        const float gamma_,
                                        const float alpha_)
    {
        cuda::SoftmaxFocalLossForward(X, T, loss, P, valid_cnt_, gamma_, alpha_);
    };

    template<typename DType>
    inline void SoftmaxFocalLossBackwardAcc(const Tensor<gpu, 3, DType> &X, // Logits; data
                                            const Tensor<gpu, 2, DType> &T, // Labels; labels
                                            const Tensor<gpu, 3, DType> &P, //softmax probability; prob
                                            const Tensor<gpu, 3, DType> &dX, // gradient out
                                            const Tensor<gpu, 2, DType> &dloss, // gradient in
                                            const Tensor<gpu, 2, DType> &buff_, // aux buff_ Tensor
                                            const int valid_cnt_,
                                            const float gamma_,
                                            const float alpha_)
    {
        cuda::SoftmaxFocalLossBackwardAcc(X, T, P, dX, dloss, buff_, valid_gt_, gamma_, alpha_);
    };

} // mshadow



namespace mxnet {
namespace op {

template<>
Operator* CreateOp<gpu>(SoftmaxFocalLossParam param, int DType) {
    Operator* op = NULL;
    MSHADOW_REAL_TYPE_SWITCH(DType, DType, {
    op = new SoftmaxFocalLossOp<gpu, DType>(param);
    });
    return op;
}

}  // namespace op
}  // namespace mxnet
